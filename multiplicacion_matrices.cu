int main() {
    int n = N;
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = n * n * sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++) {
            a[i * n + j] = i + j;
            b[i * n + j] = i * j;
        }

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 blockSize(N, N);
    dim3 gridSize((n + N - 1) / N, (n + N - 1) / N);
    matrix_mul<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++)
            printf("%d ", c[i * n + j]);
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}
