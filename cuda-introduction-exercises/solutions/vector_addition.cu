#include "hip/hip_runtime.h"
/**
   Vector addition:
   takes vectors a and b as input, computes vector sum 
   and stores output in vector c

   author: Dorothea vom Bruch (dorothea.vom.bruch@cern.ch)
   date: 05/2019

 */

#include <stdio.h>
#include <iostream>
#include <chrono>

#include "helpers.h"

using namespace std;

__constant__ int vec_size_d;

__global__ void vector_addition_kernel( int *a, int *b, int *c) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if ( index < vec_size_d ) {
    c[ index ] = a[ index ] + b[ index ];
  }
  
}

int main(int argc, char *argv[] ) {

  if ( argc != 4 ) {
    cout << "Need two arguments: size of vector, number of threads / block and device to use" << endl;
    return -1;
  }
  
  const int vec_size_h  = atoi(argv[argc-3]);
  const int n_threads = atoi(argv[argc-2]);
  const int device_id = atoi(argv[argc-1]);

  /* Chose device to use */
  CUDA_ASSERT( hipSetDevice(device_id) );
  
  cout << "Adding vectors of size " <<  vec_size_h << " with " << n_threads << " threads" << endl;

  /* Host memory for the two input vectors a and b and the output vector c */
  int *a_h = new int[vec_size_h];
  int *b_h = new int[vec_size_h];
  int *c_h = new int[vec_size_h];

  for ( int i = 0; i < vec_size_h; i++ ) {
    a_h[i] = i;
    b_h[i] = i;
    c_h[i] = 0;
  }
  
  /* Device pointers for the three vectors a, b, c */
  int *a_d, *b_d, *c_d;
  CUDA_ASSERT( hipMalloc( (void**)&a_d, vec_size_h * sizeof(int) ) );
  CUDA_ASSERT( hipMalloc( (void**)&b_d, vec_size_h * sizeof(int) ) );
  CUDA_ASSERT( hipMalloc( (void**)&c_d, vec_size_h * sizeof(int) ) );

  /* Copy vectors to device */
  CUDA_ASSERT( hipMemcpy( a_d, a_h, vec_size_h * sizeof(int), hipMemcpyHostToDevice ) );
  CUDA_ASSERT( hipMemcpy( b_d, b_h, vec_size_h * sizeof(int), hipMemcpyHostToDevice ) );
  
  CUDA_ASSERT( hipMemcpyToSymbol(HIP_SYMBOL( vec_size_d), &vec_size_h, sizeof(int) ) ); // copy to constant memory on GPU

  /* Define grid dimensions */
  int n_blocks  = vec_size_h / n_threads + (vec_size_h % n_threads != 0);
  dim3 blocks( n_blocks );
  dim3 threads(n_threads);

  std::chrono::time_point<std::chrono::system_clock> start, end;
  start = std::chrono::system_clock::now();

  /* Call kernel */
  vector_addition_kernel<<<blocks,threads>>>( a_d, b_d, c_d);

  CUDA_ASSERT( hipMemcpy( c_h, c_d, vec_size_h * sizeof(int), hipMemcpyDeviceToHost ) );

  /* Make sure GPU work is done */
  hipDeviceSynchronize();

  end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;
  
  for ( int i = 0; i < vec_size_h; i++ ) {
    cout << a_h[i] << " + " << b_h[i] << " = " << c_h[i] << endl;
  }

  cout << "Kernel duration: " << elapsed_seconds.count() << " s " << endl;
  cout << "Time per kenel: " << elapsed_seconds.count() / vec_size_h << endl;
  
  CUDA_ASSERT( hipFree( a_d ) );
  CUDA_ASSERT( hipFree( b_d ) );
  CUDA_ASSERT( hipFree( c_d ) );

  /* free host memory */
  delete [] a_h;
  delete [] b_h;
  delete [] c_h;

  
  return 0;
}
