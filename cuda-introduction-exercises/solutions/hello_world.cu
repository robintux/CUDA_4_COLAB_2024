#include "hip/hip_runtime.h"
/**
   Most simple CUDA Hello World program

   author: Dorothea vom Bruch (dorothea.vom.bruch@cern.ch)
   date: 05/2019

 */

#include <stdio.h>
#include <iostream>

#include "helpers.h"

using namespace std;

__global__ void hello_world_kernel( void ) {

  /* blockIdx.x:  Accesses index of block within grid in x direction
     threadIdx.x: Accesses index of thread within block in x direction
   */
  if ( blockIdx.x < 100 && threadIdx.x < 100 ) 
    printf("Hello World from block %u, thread %u \n", blockIdx.x, threadIdx.x);
  
}

int main( int argc, char *argv[] ) {

  if ( argc != 4 ) {
    cout << "Need three arguments: number of blocks, number of threads and device to use" << endl;
    return -1;
  }

  const int n_blocks  = atoi(argv[argc-3]);
  const int n_threads = atoi(argv[argc-2]);
  const int device_id = atoi(argv[argc-1]);

  /* Chose device to use */
  CUDA_ASSERT( hipSetDevice(device_id) );
  
  /* dim3: CUDA specific variable to declare size of grid in blocks and threads, 
     can take up to three arguments for 3-dimensional grids and blocks
  */
  dim3 blocks(n_blocks);
  dim3 threads(n_threads);

  /* Syntax to launch a kernel: 
     <<< size of grid in blocks and threads>>>
     (): any parameters to be passed to the kernel
  */
  hello_world_kernel<<<blocks,threads>>>();

  /* Blocks until all requested tasks on device were completed;
     needed for printf in kernel to work
  */
  hipDeviceSynchronize();

  return 0;
}
